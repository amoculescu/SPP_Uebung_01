#include "hip/hip_runtime.h"
//
//  kernel.cu
//
//  Created by Arya Mazaheri on 01/12/2018.
//

#include <iostream>
#include <algorithm>
#include <cmath>
#include "ppm.h"

using namespace std;

/*********** Gray Scale Filter  *********/

/**
 * Converts a given 24bpp image into 8bpp grayscale using the GPU.
 */
__global__
void cuda_grayscale(int width, int height, BYTE *image, BYTE *image_out){
    /*for (int h = 0; h < height ; h++)
    {
        int offset_out = h * width;      // 1 color per pixel
        int offset  =  offset_out * 3; // 3 colors per pixel
        for (int w = 0; w < width; w++)
        {
            BYTE *pixel = &image[offset + w * 3];
            image_out[offset_out + w] = pixel[0] * 0.0722f + // B 
            pixel[1] * 0.7152f + // G
            pixel[2] * 0.2126f;  // R ;
        }
    }*/

    int threadsPerBlock = blockDim.x * blockDim.y;
    int threadIdInBlock = threadIdx.x + blockDim.x * threadIdx.y;

    int blocksInGrid = gridDim.x * gridDim.y;
    int blockIdInGrid = blockIdx.x + gridDim.x * blockIdx.y;
    int globalThreadId = blockIdInGrid * threadsPerBlock + threadIdInBlock;
    int totalNumThreads = blocksInGrid * threadsPerBlock;

    int i = 0;
    while(totalNumThreads * i  < width * height){ 
        if(totalNumThreads * i + globalThreadId <= width * height){
            int pixelindex = (globalThreadId * 3 + totalNumThreads * 3 * i);
            BYTE *pixel = &image[pixelindex];
            image_out[globalThreadId + totalNumThreads * i] = pixel[0] * 0.0722f + // B 
            pixel[1] * 0.7152f + // G
            pixel[2] * 0.2126f;  // R 
        }           
        i++;
    }
    //TODO (9 pt): implement grayscale filter kernel
}


// 1D Gaussian kernel array values of a fixed size (make sure the number > filter size d)
//TODO: Define the cGaussian array on the constant memory (2 pt)

void cuda_updateGaussian(int r, double sd)
{
	float fGaussian[64];
	for (int i = 0; i < 2*r +1 ; i++)
	{
		float x = i - r;
		fGaussian[i] = expf(-(x*x) / (2 * sd*sd));
	}
	//TODO: Copy computed fGaussian to the cGaussian on device memory (2 pts)
	//hipMemcpyToSymbol(HIP_SYMBOL(cGaussian), fGaussian, 64 * sizeof(float). hipMemcpyDeviceToHost/* TODO */);
}

//TODO: implement cuda_gaussian() kernel (3 pts)


/*********** Bilateral Filter  *********/
// Parallel (GPU) Bilateral filter kernel
__global__ void cuda_bilateral_filter(BYTE* input, BYTE* output,
	int width, int height,
	int r, double sI, double sS)
{
	//TODO: implement bilateral filter kernel (9 pts)
}


void gpu_pipeline(const Image & input, Image & output, int r, double sI, double sS)
{
	// Events to calculate gpu run time
	hipEvent_t start, stop;
	float time;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	// GPU related variables
    BYTE *d_input = NULL;
  	BYTE *d_image_out[2] = {0}; //temporary output buffers on gpu device
	int image_size = input.cols*input.rows;
	int suggested_blockSize;   // The launch configurator returned block size 
	int suggested_minGridSize; // The minimum grid size needed to achieve the maximum occupancy for a full device launch

	// ******* Grayscale kernel launch *************

	//Creating the block size for grayscaling kernel
	hipOccupancyMaxPotentialBlockSize( &suggested_minGridSize, &suggested_blockSize, cuda_grayscale);
        
        int block_dim_x, block_dim_y;
        block_dim_x = block_dim_y = (int) sqrt(suggested_blockSize); 

        dim3 gray_block(block_dim_x, block_dim_y/* TODO */); // 2 pts

        //TODO: Calculate grid size to cover the whole image - 2 pts
        int grid_dim_x, grid_dim_y;
        grid_dim_x = fmax(input.cols / block_dim_x, suggested_minGridSize);
        grid_dim_y = fmax(input.rows / block_dim_y, suggested_minGridSize);
        dim3 gray_dim(grid_dim_x, grid_dim_y);

        // Allocate the intermediate image buffers for each step
        Image img_out(input.cols, input.rows, 1, "P5");
        cout <<  "\ninit phase"<<endl;
        for (int i = 0; i < 2; i++)
        {  
            hipError_t malloc_result = hipMalloc((void**)&d_image_out[i], image_size);
            if(hipSuccess != malloc_result)
               cout << "malloc " << i << " cuda error " << malloc_result << endl;
            else
                cout << "malloc " << i << " d_image_out successful "   << endl;
            hipError_t memset_result = hipMemset(d_image_out[i], 0xff, image_size);
            if(hipSuccess != memset_result)
                cout << "memset " << i << " cuda error  " << memset_result  << endl;
            else
                cout << "memset " << i << " d_image_out successful "  << endl;
            //TODO: allocate memory on the device (2 pts)
            //TODO: intialize allocated memory on device to zero (2 pts)
        }
        //copy input image to device
        //TODO: Allocate memory on device for input image (2 pts)
        //TODO: Copy input image into the device memory (2 pts)
        hipError_t mallocInput = hipMalloc((void**) &d_input, image_size * 3);
        if(hipSuccess != mallocInput)
            cout << "mallocInput cuda error  " << mallocInput  << endl;
        else
            cout << "malloc d_dinput successful "  << endl;

        BYTE *inputp = input.pixels;
        hipError_t copyHostToDevice =  hipMemcpy(d_input, inputp, image_size * 3, hipMemcpyHostToDevice);
        if(hipSuccess != copyHostToDevice)
            cout << "copyHostToDevice cuda error  " << hipGetErrorString(copyHostToDevice)  << endl;
        else
            cout << "copy host to device successful "  << endl;

        hipEventRecord(start, 0); // start timer
        // Convert input image to grayscale
        //TODO: Launch cuda_grayscale() (2 pts)
        cuda_grayscale<<<gray_dim, gray_block>>>(input.cols, input.rows, d_input, d_image_out[0]);
        hipEventRecord(stop, 0); // stop timer
        hipEventSynchronize(stop);

        // Calculate and print kernel run time
        hipEventElapsedTime(&time, start, stop);
        cout << "GPU Grayscaling time: " << time << " (ms)\n";
        cout << "Launched blocks of size " << gray_block.x * gray_block.y << endl;
    
        //TODO: transfer image from device to the main memory for saving onto the disk (2 pts)
        hipError_t copyDeviceToHost  = hipMemcpy(img_out.pixels, d_image_out[0],  image_size, hipMemcpyDeviceToHost);
        if(hipSuccess != copyDeviceToHost)
            cout << "copyDeviceToHost cuda error  " << copyDeviceToHost  << endl;
        else   
            cout << "copy device to host successful "  << endl;

        savePPM(img_out, "image_gpu_gray.ppm");
        hipFree(d_input);
        hipFree(d_image_out);


	// ******* Bilateral filter kernel launch *************
	
	//Creating the block size for grayscaling kernel
	hipOccupancyMaxPotentialBlockSize( &suggested_minGridSize, &suggested_blockSize, cuda_bilateral_filter); 
        
        block_dim_x = block_dim_y = (int) sqrt(suggested_blockSize); 

        dim3 bilateral_block(/* TODO */); // 2 pts

        //TODO: Calculate grid size to cover the whole image - 2pts

        // Create gaussain 1d array
        cuda_updateGaussian(r,sS);

        hipEventRecord(start, 0); // start timer
	//TODO: Launch cuda_bilateral_filter() (2 pts)
        hipEventRecord(stop, 0); // stop timer
        hipEventSynchronize(stop);

        // Calculate and print kernel run time
        hipEventElapsedTime(&time, start, stop);
        cout << "\nGPU Bilateral Filter time: " << time << " (ms)\n";
       // cout << "Launched blocks of size " << bilateral_block.x * bilateral_block.y << endl;

        // Copy output from device to host
	//TODO: transfer image from device to the main memory for saving onto the disk (2 pts)


        // ************** Finalization, cleaning up ************

        // Free GPU variables
	//TODO: Free device allocated memory (3 pts)
}
