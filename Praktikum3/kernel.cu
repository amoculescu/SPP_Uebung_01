#include "hip/hip_runtime.h"
//
//  kernel.cu
//
//  Created by Arya Mazaheri on 01/12/2018.
//

#include <iostream>
#include <algorithm>
#include <cmath>
#include "ppm.h"

using namespace std;

/*********** Gray Scale Filter  *********/

/**
 * Converts a given 24bpp image into 8bpp grayscale using the GPU.
 */
__global__
void cuda_grayscale(int width, int height, BYTE *image, BYTE *image_out){
    int threadsPerBlock = blockDim.x * blockDim.y;
    int threadIdInBlock = threadIdx.x + blockDim.x * threadIdx.y;

    int blocksInGrid = gridDim.x * gridDim.y;
    int blockIdInGrid = blockIdx.x + gridDim.x * blockIdx.y;
    int globalThreadId = blockIdInGrid * threadsPerBlock + threadIdInBlock;
    int totalNumThreads = blocksInGrid * threadsPerBlock;

    int i = 0;
    while(totalNumThreads * i  < width * height){ 
        if(totalNumThreads * i + globalThreadId <= width * height){
            int pixelindex = (globalThreadId * 3 + totalNumThreads * 3 * i);
            BYTE *pixel = &image[pixelindex];
            image_out[globalThreadId + totalNumThreads * i] = pixel[0] * 0.0722f + // B 
            pixel[1] * 0.7152f + // G
            pixel[2] * 0.2126f;  // R 
        }           
        i++;
    }
}


// 1D Gaussian kernel array values of a fixed size (make sure the number > filter size d)
__constant__ float cGaussian[64];
void cuda_updateGaussian(int r, double sd)
{
	float fGaussian[64];
	for (int i = 0; i < 2*r +1 ; i++)
	{
		float x = i - r;
		fGaussian[i] = expf(-(x*x) / (2 * sd*sd));
	}
    hipError_t copyHostToDeviceSymbol = hipMemcpyToSymbol(HIP_SYMBOL(cGaussian), fGaussian, 64 * sizeof(float), 0, hipMemcpyHostToDevice);
    if(hipSuccess != copyHostToDeviceSymbol)
        cout << "copy to Symbol on device cuda error  " << copyHostToDeviceSymbol  << endl;
    else   
        cout << "copy to Symbol on device successful "  << endl;
}

//TODO: implement cuda_gaussian() kernel (3 pts)
__device__ double cuda_gaussian(float x, double sigma){
	return expf(-(powf(x, 2)) / (2 * powf(sigma, 2)));
}

/*********** Bilateral Filter  *********/
// Parallel (GPU) Bilateral filter kernel
__global__ void cuda_bilateral_filter(BYTE* input, BYTE* output,
	int width, int height,
	int r, double sI, double sS)
{
    // for(int h = 0; h < height; h++){
	// 	for(int w = 0; w < width; w++){
	// 		double iFiltered = 0;
	// 		double wP = 0;
	// 		// Get the centre pixel value
	// 		unsigned char centrePx = input[h*width+w];
	// 		// Iterate through filter size from centre pixel
	// 		for (int dy = -r; dy <= r; dy++) {
	// 			int neighborY = h+dy;
	// 			if (neighborY < 0)
    //                 neighborY = 0;
    //             else if (neighborY >= height)
    //                 neighborY = height - 1;
	// 			for (int dx = -r; dx <= r; dx++) {
	// 				int neighborX = w+dx;
	// 				if (neighborX < 0)
	//                     neighborX = 0;
	//                 else if (neighborX >= width)
	//                     neighborX = width - 1;
	// 				// Get the current pixel; value
	// 				unsigned char currPx = input[neighborY*width+neighborX];
	// 				// Weight = 1D Gaussian(x_axis) * 1D Gaussian(y_axis) * Gaussian(Range or Intensity difference)
	// 				double w = (fGaussian[dy + r] * fGaussian[dx + r]) * cpu_gaussian(centrePx - currPx, sI);
	// 				iFiltered += w * currPx;
	// 				wP += w;				
	// 			}
	// 		}
	// 		output[h*width + w] = iFiltered / wP;
	// 	}
    // }
    int threadsPerBlock = blockDim.x * blockDim.y;
    int threadIdInBlock = threadIdx.x + blockDim.x * threadIdx.y;

    int blocksInGrid = gridDim.x * gridDim.y;
    int blockIdInGrid = blockIdx.x + gridDim.x * blockIdx.y;
    int globalThreadId = blockIdInGrid * threadsPerBlock + threadIdInBlock;
    int totalNumThreads = blocksInGrid * threadsPerBlock;

    int i = 0;
    int neighborX;
    int neighborY;
    while(totalNumThreads * i  < width * height){ 
        double iFiltered = 0;
        double wP = 0;
        int pixelindex = (globalThreadId + totalNumThreads * i);
        unsigned char centrePx = input[pixelindex];
        for (int dy = -r; dy < 0; dy++){
            neighborY = fmaxf(0, pixelindex - width * dy);
            for(int dx = -r; dx < 0; dy++){
                neighborX = fmaxf(0, dx);
                unsigned char currPx = input[neighborY - neighborX];
                double w = (cGaussian[dy + r] * cGaussian[dx + r]) * cuda_gaussian(centrePx - currPx, sI);
                iFiltered += w * currPx;
                wP += w;
            }
            for(int dx = 1; dx <= r; dx++){
                neighborX = fminf(width, dx);
                unsigned char currPx = input[neighborY - neighborX];
                double w = (cGaussian[dy + r] * cGaussian[dx + r]) * cuda_gaussian(centrePx - currPx, sI);
                iFiltered += w * currPx;
                wP += w;
            }
        }
        for(int dy = 1; dy <= r; dy++){ 
            neighborY = fminf(height * width, pixelindex + width * dy);
            for(int dx = -r; dx < 0; dy++){
                neighborX = fmaxf(0, dx);
                unsigned char currPx = input[neighborY - neighborX];
                double w = (cGaussian[dy + r] * cGaussian[dx + r]) * cuda_gaussian(centrePx - currPx, sI);
                iFiltered += w * currPx;
                wP += w;
            }
            for(int dx = 1; dx <= r; dx++){
                neighborX = fminf(width, dx);
                unsigned char currPx = input[neighborY - neighborX];
                double w = (cGaussian[dy + r] * cGaussian[dx + r]) * cuda_gaussian(centrePx - currPx, sI);
                iFiltered += w * currPx;
                wP += w;
            }
        }
        output[pixelindex] = iFiltered / wP;
    }
}


void gpu_pipeline(const Image & input, Image & output, int r, double sI, double sS)
{
	// Events to calculate gpu run time
	hipEvent_t start, stop;
	float time;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	// GPU related variables
    BYTE *d_input = NULL;
  	BYTE *d_image_out[2] = {0}; //temporary output buffers on gpu device
	int image_size = input.cols*input.rows;
	int suggested_blockSize;   // The launch configurator returned block size 
	int suggested_minGridSize; // The minimum grid size needed to achieve the maximum occupancy for a full device launch

	// ******* Grayscale kernel launch *************

	//Creating the block size for grayscaling kernel
	hipOccupancyMaxPotentialBlockSize( &suggested_minGridSize, &suggested_blockSize, cuda_grayscale);
        
        int block_dim_x, block_dim_y;
        block_dim_x = block_dim_y = (int) sqrt(suggested_blockSize); 

        dim3 gray_block(block_dim_x, block_dim_y); // 2 pts

        int grid_dim_x, grid_dim_y;
        grid_dim_x = fmax(input.cols / block_dim_x, suggested_minGridSize);
        grid_dim_y = fmax(input.rows / block_dim_y, suggested_minGridSize);
        dim3 gray_grid(grid_dim_x, grid_dim_y);

        // Allocate the intermediate image buffers for each step
        Image img_out(input.cols, input.rows, 1, "P5");
        cout <<  "\ninit phase"<<endl;
        for (int i = 0; i < 2; i++)
        {  
            hipError_t malloc_result = hipMalloc((void**)&d_image_out[i], image_size);
            if(hipSuccess != malloc_result)
               cout << "malloc " << i << " cuda error " << malloc_result << endl;
            else
                cout << "malloc d_image_out[" << i << "] successful "   << endl;
            hipError_t memset_result = hipMemset(d_image_out[i], 0xff, image_size);
            if(hipSuccess != memset_result)
                cout << "memset " << i << " cuda error  " << memset_result  << endl;
            else
                cout << "memset d_image_out[" << i << "] successful "  << endl;
         }
        //copy input image to device
        hipError_t mallocInput = hipMalloc((void**) &d_input, image_size * 3);
        if(hipSuccess != mallocInput)
            cout << "malloc Input cuda error  " << mallocInput  << endl;
        else
            cout << "malloc d_dinput successful "  << endl;

        BYTE *inputp = input.pixels;
        hipError_t copyHostToDevice =  hipMemcpy(d_input, inputp, image_size * 3, hipMemcpyHostToDevice);
        if(hipSuccess != copyHostToDevice)
            cout << "copyHostToDevice cuda error  " << hipGetErrorString(copyHostToDevice)  << endl;
        else
            cout << "copy input to device successful "  << endl;

        hipEventRecord(start, 0); // start timer
        // Convert input image to grayscale
        cuda_grayscale<<<gray_grid, gray_block>>>(input.cols, input.rows, d_input, d_image_out[0]);
        hipEventRecord(stop, 0); // stop timer
        hipEventSynchronize(stop);

        // Calculate and print kernel run time
        hipEventElapsedTime(&time, start, stop);
        cout << "GPU Grayscaling time: " << time << " (ms)\n";
        cout << "Launched blocks of size " << gray_block.x * gray_block.y << endl;
    
         hipError_t copyDeviceToHost  = hipMemcpy(img_out.pixels, d_image_out[0],  image_size, hipMemcpyDeviceToHost);
        if(hipSuccess != copyDeviceToHost)
            cout << "copyDeviceToHost cuda error  " << copyDeviceToHost  << endl;
        else   
            cout << "copy device to host successful "  << endl;

        savePPM(img_out, "image_gpu_gray.ppm");
        hipFree(d_input);
        hipFree(d_image_out);


	// ******* Bilateral filter kernel launch *************
	
	//Creating the block size for grayscaling kernel
	hipOccupancyMaxPotentialBlockSize( &suggested_minGridSize, &suggested_blockSize, cuda_bilateral_filter); 
        
        block_dim_x = block_dim_y = (int) sqrt(suggested_blockSize); 

        dim3 bilateral_block(block_dim_x, block_dim_y); // 2 pts

        //TODO: Calculate grid size to cover the whole image - 2pts
        grid_dim_x = fmax(input.cols / block_dim_x, suggested_minGridSize);
        grid_dim_y = fmax(input.rows / block_dim_y, suggested_minGridSize);
        dim3 bilateral_grid(grid_dim_x, grid_dim_y);
        // Create gaussian 1d array
        cuda_updateGaussian(r,sS);

        hipEventRecord(start, 0); // start timer
    //TODO: Launch cuda_bilateral_filter() (2 pts)
        cuda_bilateral_filter<<<bilateral_grid, bilateral_block>>>
        (d_image_out[0], d_image_out[1], input.cols, input.rows, r, sI, sS);
        hipEventRecord(stop, 0); // stop timer
        hipEventSynchronize(stop);

        // Calculate and print kernel run time
        hipEventElapsedTime(&time, start, stop);
        cout << "\nGPU Bilateral Filter time: " << time << " (ms)\n";
        cout << "Launched blocks of size " << bilateral_block.x * bilateral_block.y << endl;

        // Copy output from device to host

	//TODO: transfer image from device to the main memory for saving onto the disk (2 pts)

    hipError_t copyDeviceToHostBilateral  = hipMemcpy(img_out.pixels, d_image_out[1],  image_size, hipMemcpyDeviceToHost);
    if(hipSuccess != copyDeviceToHostBilateral)
        cout << "copyDeviceToHostBilateral cuda error  " << copyDeviceToHostBilateral  << endl;
    else   
        cout << "copy bilateral from device to host successful "  << endl;

    savePPM(img_out, "image_gpu.ppm");

        // ************** Finalization, cleaning up ************

        // Free GPU variables
	//TODO: Free device allocated memory (3 pts)
}
